#include "hip/hip_runtime.h"
#include <buffer_manager.h>

#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <stdio.h>

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
                  file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__global__ void fade_buffer_kernel(unsigned char* buff, int width, int height){

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    
    // printf("X: %d  | Y: %d\n", x, y);

    // std::cout << "X: " << x << "Y: " << y << std::endl;

    if(y > height || x > width) return;

    int pixel_index = y * width * 3 + x * 3;

    buff[pixel_index] = (unsigned char)(255.99f * (float(x) / float(width)));
    buff[pixel_index + 1] = (unsigned char)(255.99f * (float(y) / float(height)));
    buff[pixel_index + 2] = 60;

}

void save_ppm(const char *path, unsigned char *buff, int width, int height){
    std::ofstream fp;
    fp.open(path);

    fp << "P3\n" << width << " " << height << "\n255\n";
    for (int j = height-1; j >= 0; j--) {
        for (int i = 0; i < width; i++) {

            // std::cout << "X: " << i << "Y: " << j << std::endl;
            // printf("X: %d  | Y: %d\n", i, j);
            int pixel_index = j * 3 * width + i * 3;

            unsigned char r = buff[pixel_index];
            unsigned char g = buff[pixel_index + 1];
            unsigned char b = buff[pixel_index + 2];
   
            fp << int(r) << " " << int(g) << " " << int(b) << "\n";
        }
    }

    fp.close();
}

int main(int argc, char const *argv[])
{

    const int width = 800, height = 600;
    const int num_thread_per_block = 100;

    dim3 blocks(width / num_thread_per_block + 1, height / num_thread_per_block + 1);
    dim3 threads(num_thread_per_block, num_thread_per_block);

    unsigned char* buff;
    checkCudaErrors(hipMalloc(&buff, width * height * 3));
    unsigned char *outbuff = (unsigned char *)malloc(width * height * 3);

    fade_buffer_kernel<<<blocks, threads>>>(buff, width, height);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hipMemcpy(outbuff, buff, width * height * 3, hipMemcpyDeviceToHost);
    save_ppm("testimage.ppm", outbuff, 800, 600);

    hipFree(buff);
    free(outbuff);

    return 0;
}
